#include "hip/hip_runtime.h"
#include "test_add.h"

__global__ void add(int *a, int *b, int *c){
    *c = *a + *b;
}

extern "C" {
int test_add(void) {
    int a, b, c; // host copies of a, b, c
    int *d_a, *d_b, *d_c; // device copies of a, b, c
    int size = sizeof(int);
    // Allocate space for device copies of a, b, c
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);
    // Setup input values
    a = 2;
    b = 7;

    // Copy inputs to device
    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);
    // Launch add() kernel on GPU
    add<<<1,1>>>(d_a, d_b, d_c);
    // Copy result back to host
    hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
    // Cleanup
    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    return 0;
}
}
